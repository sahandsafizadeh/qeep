#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <math.h>

#include "common.h"

/* ----- kernel launch ----- */

const int MAX_THREADS_PER_BLOCK_X = 512;
const int MAX_BLOCKS_PER_GRID_X = 65535;

LaunchParams launchParams(size_t n)
{
    int threads = MAX_THREADS_PER_BLOCK_X;
    int blocks = (n + threads - 1) / threads;
    blocks = min(blocks, MAX_BLOCKS_PER_GRID_X);
    return (LaunchParams){blocks, threads};
}

/* ----- error handling ----- */

void handleCudaError(hipError_t err)
{
    if (err != hipSuccess)
    {
        printf("%s: %s\n",
               hipGetErrorName(err),
               hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
}