#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <math.h>

#include "common.h"

/* ----- kernel launch ----- */

LaunchParams launchParams(size_t n)
{
    unsigned int threads = MAX_THREADS_PER_BLOCK_X;
    unsigned int blocks = (n + threads - 1) / threads;
    blocks = min(blocks, MAX_BLOCKS_PER_GRID_X);
    return (LaunchParams){blocks, threads};
}

/* ----- error handling ----- */

void handleCudaError(hipError_t err)
{
    if (err != hipSuccess)
    {
        printf("%s: %s\n",
               hipGetErrorName(err),
               hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
}