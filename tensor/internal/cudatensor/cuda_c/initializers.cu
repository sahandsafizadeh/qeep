#include "hip/hip_runtime.h"
#include "common.h"

/* ----- device functions ----- */

__global__ void fillDataWithValue(double *data, size_t n, double value)
{
    const int tpos = getThreadPosition();
    const int gstep = getGridStepSize();

    for (size_t i = tpos; i < n; i += gstep)
    {
        data[i] = value;
    }
}

/* ----- API functions ----- */

double *Full(size_t n, double value)
{
    double *dev_data;
    hipMalloc(&dev_data, n * sizeof(double));

    fillDataWithValue<<<BLOCKS, THREADS>>>(dev_data, n, value);
    hipDeviceSynchronize();

    return dev_data;
}

void FreeCUDAMemory(double *dev_data)
{
    hipFree(dev_data);
}