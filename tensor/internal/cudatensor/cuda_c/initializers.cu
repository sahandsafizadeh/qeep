#include "hip/hip_runtime.h"
#include <time.h>
#include <hiprand/hiprand_kernel.h>

#include "types.h"
#include "common.cuh"
#include "devcommon.cuh"

/* ----- helper functions ----- */

inline unsigned long long timeSeed()
{
    return (unsigned long long)time(NULL);
}

/* ----- device functions ----- */

__device__ int toConcatenatedPosition(int lnpos_src, DimArr rcp_src, DimArr rcp_dst, int dim, int mrg)
{
    int lnpos_dst;
    DimArr index_src;
    DimArr index_dst;

    index_src = decode(lnpos_src, rcp_src);

    index_dst = index_src;
    index_dst.arr[dim] += mrg;

    lnpos_dst = encode(index_dst, rcp_dst);

    return lnpos_dst;
}

__global__ void fillConst(CudaData dst, double value)
{
    const unsigned int tpos = threadPosition();
    const unsigned int stride = totalThreads();

    for (size_t i = tpos; i < dst.size; i += stride)
    {
        dst.arr[i] = value;
    }
}

__global__ void fillEye(CudaData dst, size_t d)
{
    const unsigned int tpos = threadPosition();
    const unsigned int stride = totalThreads();

    for (size_t i = tpos; i < dst.size; i += stride)
    {
        dst.arr[i] = i % (d + 1) == 0 ? 1. : 0.;
    }
}

__global__ void fillRandU(CudaData dst, double l, double u, unsigned long long seed)
{
    const unsigned int tpos = threadPosition();
    const unsigned int stride = totalThreads();

    hiprandStatePhilox4_32_10_t state;
    hiprand_init(seed, tpos, 0, &state);

    for (size_t i = tpos; i < dst.size; i += stride)
    {
        double randu_0_1 = hiprand_uniform_double(&state);
        dst.arr[i] = l + (u - l) * randu_0_1;
    }
}

__global__ void fillRandN(CudaData dst, double u, double s, unsigned long long seed)
{
    const unsigned int tpos = threadPosition();
    const unsigned int stride = totalThreads();

    hiprandStatePhilox4_32_10_t state;
    hiprand_init(seed, tpos, 0, &state);

    for (size_t i = tpos; i < dst.size; i += stride)
    {
        double randn_0_1 = hiprand_normal_double(&state);
        dst.arr[i] = u + s * randn_0_1;
    }
}

__global__ void fillConcat(CudaData dst, CudaData src, DimArr rcp_dst, DimArr rcp_src, int dim, int mrg)
{
    const unsigned int tpos = threadPosition();
    const unsigned int stride = totalThreads();

    for (size_t i = tpos; i < src.size; i += stride)
    {
        int lnpos_src = i;
        int lnpos_dst = toConcatenatedPosition(lnpos_src, rcp_src, rcp_dst, dim, mrg);

        dst.arr[lnpos_dst] = src.arr[lnpos_src];
    }
}

/* ----- API functions ----- */

extern "C"
{
    double *Full(size_t n, double value);
    double *Eye(size_t n, size_t d);
    double *RandU(size_t n, double l, double u);
    double *RandN(size_t n, double u, double s);
    double *Of(size_t n, double *input_data);
    double *Concat(CudaData srcs[], DimArr dims_srcs[], size_t size, int dim, DimArr dims_dst);
}

double *Full(size_t n, double value)
{
    CudaData dst = (CudaData){NULL, n};
    handleCudaError(
        hipMalloc(&dst.arr, dst.size * sizeof(double)));

    LaunchParams lps = launchParams(dst.size);

    fillConst<<<lps.blockSize, lps.threadSize>>>(dst, value);

    handleCudaError(
        hipGetLastError());
    handleCudaError(
        hipDeviceSynchronize());

    return dst.arr;
}

double *Eye(size_t n, size_t d)
{
    CudaData dst = (CudaData){NULL, n};
    handleCudaError(
        hipMalloc(&dst.arr, dst.size * sizeof(double)));

    LaunchParams lps = launchParams(dst.size);

    fillEye<<<lps.blockSize, lps.threadSize>>>(dst, d);

    handleCudaError(
        hipGetLastError());
    handleCudaError(
        hipDeviceSynchronize());

    return dst.arr;
}

double *RandU(size_t n, double l, double u)
{
    CudaData dst = (CudaData){NULL, n};
    handleCudaError(
        hipMalloc(&dst.arr, dst.size * sizeof(double)));

    unsigned long long seed = timeSeed();

    LaunchParams lps = launchParams(dst.size);

    fillRandU<<<lps.blockSize, lps.threadSize>>>(dst, l, u, seed);

    handleCudaError(
        hipGetLastError());
    handleCudaError(
        hipDeviceSynchronize());

    return dst.arr;
}

double *RandN(size_t n, double u, double s)
{
    CudaData dst = (CudaData){NULL, n};
    handleCudaError(
        hipMalloc(&dst.arr, dst.size * sizeof(double)));

    unsigned long long seed = timeSeed();

    LaunchParams lps = launchParams(dst.size);

    fillRandN<<<lps.blockSize, lps.threadSize>>>(dst, u, s, seed);

    handleCudaError(
        hipGetLastError());
    handleCudaError(
        hipDeviceSynchronize());

    return dst.arr;
}

double *Of(size_t n, double *input_data)
{
    double *dst;
    handleCudaError(
        hipMalloc(&dst, n * sizeof(double)));

    handleCudaError(
        hipMemcpy(
            dst,
            input_data,
            n * sizeof(double),
            hipMemcpyHostToDevice));

    return dst;
}

double *Concat(CudaData srcs[], DimArr dims_srcs[], size_t size, int dim, DimArr dims_dst)
{
    size_t n = elemcnt(dims_dst);
    DimArr rcp_dst = rcumprod(dims_dst);

    CudaData dst = (CudaData){NULL, n};
    handleCudaError(
        hipMalloc(&dst.arr, dst.size * sizeof(double)));

    int mrg = 0;
    for (size_t i = 0; i < size; i++)
    {
        CudaData src = srcs[i];
        DimArr dims_src = dims_srcs[i];
        DimArr rcp_src = rcumprod(dims_src);

        LaunchParams lps = launchParams(src.size);

        fillConcat<<<lps.blockSize, lps.threadSize>>>(dst, src, rcp_dst, rcp_src, dim, mrg);

        handleCudaError(
            hipGetLastError());
        handleCudaError(
            hipDeviceSynchronize());

        mrg += dims_src.arr[dim];
    }

    return dst.arr;
}