#include "types.h"
#include "common.h"

/* ----- API functions ----- */

extern "C"
{
    double *Reshape(CudaData src);
}

double *Reshape(CudaData src)
{
    CudaData dst = (CudaData){NULL, src.size};
    handleCudaError(
        hipMalloc(&dst.arr, dst.size * sizeof(double)));
    handleCudaError(
        hipMemcpy(
            dst.arr,
            src.arr,
            src.size * sizeof(double),
            hipMemcpyDeviceToDevice));

    return dst.arr;
}